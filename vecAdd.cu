#include <hip/hip_runtime.h>


__global__ 
void vecAddKernel(float* A, float* B, float* C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

extern "C" void vecAdd(float* A_h, float* B_h, float* C_h, int n){
    float* A_d, *B_d, *C_d;
    int size = n * sizeof(float);

    hipMalloc((void **) &A_d, size);
    hipMalloc((void **) &B_d, size);
    hipMalloc((void **) &C_d, size);

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);
    
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    vecAddKernel<<<numBlocks, blockSize>>>(A_d, B_d, C_d, n);

    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}
