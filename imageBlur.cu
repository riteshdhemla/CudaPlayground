
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void imageBlurKernel(float* A, float* T, int w, int h)
{
    // w : width , number of columns
    // h : height , number of rows
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    
    int BLUR_SIZE = 1; // BLUR_MATRIX = 3 x 3
    float sum_pixels = 0.0f;
    int pixel_count = 0;
    int colIdx, rowIdx;
    
    if (col < w && row < h)
    {
        for(int blurRow=-BLUR_SIZE; blurRow <= BLUR_SIZE; blurRow++)
        {
            for(int blurCol=-BLUR_SIZE; blurCol <= BLUR_SIZE; blurCol++)
            {
                colIdx = col + blurCol;
                rowIdx = row + blurRow;
                if (colIdx >= 0 && rowIdx >= 0 && colIdx < w && rowIdx < h)
                {
                    sum_pixels += A[rowIdx * w + colIdx];
                    ++pixel_count;
                }
            }
        }
        T[row * w + col] = sum_pixels / pixel_count;
    }
}

extern "C" void imageBlur(float* A_h, float* T_h, int w, int h)
{
    float* A_d;
    float* T_d;
    int memSize = sizeof(float) * w * h;
    hipError_t err;
    
    err = hipMalloc((void **) &A_d, memSize);

    if (err != hipSuccess)
    {
        printf("Error allocating device memory for A_d");
        return;
    }

    err = hipMalloc((void **) &T_d, memSize);
    if (err != hipSuccess)
    {
        printf("Error allocating device memory for T_d");
        return;
    }
    hipMemcpy(A_d, A_h, memSize, hipMemcpyHostToDevice);
  
    dim3 dimGrid(ceil(w/16.0), ceil(h/16.0), 1);
    dim3 dimBlock(16, 16, 1);
    imageBlurKernel<<<dimGrid,dimBlock>>>(A_d, T_d, w, h);

    hipMemcpy(T_h, T_d, memSize, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(T_d);
}